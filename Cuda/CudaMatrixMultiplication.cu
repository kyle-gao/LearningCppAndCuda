
#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;

#define N 8


void cpuMatMul(float A[N][N], float B[N][N], float out[N][N]);

__global__ void gpuMatMul(float *A, float *B, float *out, int W);

int main(){


    float A[N][N];
    float B[N][N];
    float C[N][N];
    
    float* pA;
    float* pB;
    float* pC;

    pA = (float*)malloc(sizeof(float) *N* N);
    pB = (float*)malloc(sizeof(float) *N* N);
    pC = (float*)malloc(sizeof(float) *N* N);


    float *dA, *dB, *dC;


    // Allocate memory
    //a = (float*)malloc(sizeof(float) * N);
    //b = (float*)malloc(sizeof(float) * N);
    //out = (float*)malloc(sizeof(float) * N);
    
    //initialize the arrays
    for (int k = 0;k < N; k++) {
        for (int m = 0; m < N;m++) {
            A[m][k] = 2*(float)m + 3*(float)k;
            pA[m*N + k] = A[m][k];
            B[m][k] = 3*(float)m + 4*(float)k;
            pB[m*N + k] = B[m][k];
        }
    }

    cpuMatMul(A, B, C);





    hipMalloc((void**)&dA, sizeof(float) * N * N); //void** is a pointer to a pointer of unspecified type
    hipMalloc((void**)&dB, sizeof(float) * N * N);
    hipMalloc((void**)&dC, sizeof(float) * N * N);


    hipMemcpy(dA, pA, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(dB, pB, sizeof(float) * N * N, hipMemcpyHostToDevice);


    //using 2d indexing of threads and blocks
    dim3 threads(4, 4);
    dim3 blocks(2, 2);

    //We must have threads * blocks = (N,N)
    gpuMatMul <<<blocks, threads >>>(dA, dB, dC, N);
    hipMemcpy(pC, dC, sizeof(float) * N * N, hipMemcpyDeviceToHost);


    cout << endl;
    for (int m = 0; m < N;m++) {
        for (int n = 0; n < N;n++) {
            cout << C[m][n] << " ";
        }
        cout << endl;
    }
    cout << endl;

    for (int m = 0; m < N;m++) {
        for (int n = 0; n < N;n++) {
            cout << pC[m*N+n] << " ";
        }
        cout << endl;
    }
    cout << endl;

    //free memory
    hipFree(dA);hipFree(dB);hipFree(dC);
    free(pA);free(pB);free(pC);
    return 0;
    }

void cpuMatMul(float A[N][N], float B[N][N], float out[N][N]) {
    float sum = 0.0;
    for (int i = 0; i < N; i++) {
        for (int j = 0;j < N; j++) {
            sum = 0.0;
            for (int k = 0; k < N;k++) {
                sum = sum + A[i][k] * B[k][j];
            }
            out[i][j] = sum;
        }
    }
}

__global__ void gpuMatMul(float *A, float *B, float *out, int W) {
    //element(x, y) can be addressed as : x * width + y
    //eg for a 4x4 matrix, m(1,1) = 5 = 1*4+1

    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;
    if (row < W && col < W) {
        //prevents us from going out of bound in case we launch more threads/blocks
        //than necessary
        for (int k = 0;k < W;k++) {
            //sum = sum + A[row][k] + B[k][col]
            sum += A[row * W + k] * B[k * W + col];
        }
        out[row*W+col] = sum;
    }
}