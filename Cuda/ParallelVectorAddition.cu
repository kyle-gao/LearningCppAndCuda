#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h> //printf() comes from here
#include <iostream>
using namespace std;


#define N 100
#define THREADS_PER_BLOCK 10;

__global__ void cuda_hello() { //__global__ declares the function to run on GPU
    printf("Hello, I'm on the gpu");
}

__global__ void vectoradd(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n;i++)
        out[i] = a[i] + b[i];
}

__global__ void parallel_addB(float* out, float* a, float* b) {
    //block parallel function
    out[threadIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void parallel_addT(float* out, float* a, float* b) {
    //thread parallel function
    out[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void parallel_add(float* out, float* a, float* b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    //each block has blockDim threads. 
    //eg. if each block has 5 threads, index 13 is the 4th index of block 3
    out[index] = a[index] + b[index];

}

int main(){
    cuda_hello <<<1, 1 >>>  (); 
    //<<< M , T >>> indicates launching in parallel M blocks each with T threads


    float* a, * b, * out;

    // Allocate memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    
    //initialize the array
    for (int i = 0;i < N; i++) {
        //recall that array[index]=*(array+index)
        a[i]= 1.0f;
        b[i] = 2.0f;
    }
    cout << a[3] << endl;


    //vectoradd(out, a, b, N)<<<1,1>>>; 
    //does not work; a,b,out are declared on cpu and are in RAM
    //they need to be in GPU memory

    //can use hipMalloc/hipFree to allocate on GPU memory
    //can use hipMemcpy to transfer values from host to device memory
    float *d_a;
    float *d_b;
    float *d_out;

    
    hipMalloc((void**)&d_a, sizeof(float) * N); //void** is a pointer to a pointer of unspecified type
    hipMalloc((void**)&d_b, sizeof(float) * N); 
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);


    vectoradd << <1, 1 >> > (d_out, d_a, d_b,N);
    hipMemcpy (out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
   
   


    cout <<endl<<out[10]<<endl;

    parallel_addB << <N, 1 >> > (d_out, d_a, d_b);  //run on N blocks in parallel
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    cout << endl << out[10] << endl;


    parallel_addT << <1, N >> > (d_out, d_a, d_b);  //run on N threads in parallel
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    cout << endl << out[10] << endl;


    parallel_add << <10, 10>> > (d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    cout << endl << out[10] << endl;






    //remember to unallocate
    hipFree(d_a);hipFree(d_b);hipFree(d_out);
    free(a);free(b);free(out);
    return 0;
    }

